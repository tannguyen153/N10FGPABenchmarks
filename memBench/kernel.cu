
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#define TYPE float
#define WARP_SIZE 32
#define THREADS_PER_BLOCK 512
#define NWARPS THREADS_PER_BLOCK/WARP_SIZE
#define ENABLE_READ 0
#define ENABLE_READWRITE 1
#define TEST_RANDOMICITY 0
#if TEST_RANDOMICITY==1
#define RANDOM_SEGMENTS 1
#define RANDOM_ELEMENTS 0
#endif

#define TEST_SCRATCHPAD 1
#if TEST_SCRATCHPAD==1
#define ONCHIP_BUFFERSIZE 512
#endif


#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#if ENABLE_READ==1
__inline__ __device__
TYPE warpSumReduction(TYPE val) {
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2)
        val += __shfl_down(val, offset);
    return val;
}
#endif

__global__ void kernel(TYPE *mat, TYPE *val, unsigned long num_thread_blocks,unsigned long num_segments, unsigned long num_elements, int ntimes) 
{
    static __shared__ TYPE psum[32];
    int lane = threadIdx.x % WARP_SIZE;
    int wid = threadIdx.x / WARP_SIZE;
    int dataBlockSize= num_segments*num_elements;
    int gidx = blockIdx.x*dataBlockSize;
#if TEST_SCRATCHPAD==1
    __shared__ TYPE data[NWARPS][ONCHIP_BUFFERSIZE];
    for (int i= lane; i < ONCHIP_BUFFERSIZE; i+=WARP_SIZE)
	data[wid][i]=0;
#endif

    TYPE tmp_val=0;
#if TEST_RANDOMICITY==1
    for(unsigned long j0=wid; j0<num_segments; j0+=NWARPS)
    {
        unsigned long j= (j0+ wid*NWARPS)%num_segments;
#else
    for(unsigned long j=wid; j<num_segments; j+=NWARPS)
    {
#endif
        unsigned long start=gidx+j*num_elements;
        unsigned long stop= start+num_elements;
	#if TEST_SCRATCHPAD==1
        for (unsigned long i = start; i < stop; i+=ONCHIP_BUFFERSIZE) {
            //load part of the segment from HBM to scratchpad
	    for (unsigned long i0 = lane; i0 < ONCHIP_BUFFERSIZE && i+i0<stop; i0+=WARP_SIZE)
   	        data[wid][i0]= mat[i+i0];
	    for(int r=0; r<ntimes; r++){
		for (unsigned long i0 = lane; i0 < ONCHIP_BUFFERSIZE; i0+=WARP_SIZE){
	            data[wid][i0]+=1.0; 
		}
	    }
	    //store the data back to HBM
	    for (unsigned long i0 = lane; i0 < ONCHIP_BUFFERSIZE && i+i0<stop; i0+=WARP_SIZE)
	    {
		mat[i+i0]= data[wid][i0];
//		if(wid==0 && lane==0)printf("Final value at %ul is %f\n", i+i0, data[wid][i0]);
	    }
	}
	#else
        for(int r=0; r<ntimes; r++)
            for (unsigned long i = start+ lane; i < stop; i+=WARP_SIZE) {
   	        #if ENABLE_READ==1
	        tmp_val += mat[i]; 
	        #elif ENABLE_READWRITE==1
	        tmp_val =  mat[i]+1.0;
	        mat[i]= tmp_val;
	        #endif
        }
	#endif
    }

#if ENABLE_READ==1
    tmp_val = warpSumReduction(tmp_val);
    if (lane==0) psum[wid]=tmp_val;
    __syncthreads();
    tmp_val = (threadIdx.x < blockDim.x / WARP_SIZE) ? psum[lane] : 0;
    if (wid==0){
        tmp_val = warpSumReduction(tmp_val);
        if(threadIdx.x==0)atomicAdd(val, tmp_val);
    }
#elif ENABLE_READWRITE==1 
#endif
}


//we don't want to set VEC_LENGTH to 1, as it will trigger the optimizer that caches data in register instead of shared memory in some certain cases
#define VEC_LENGTH 2
#define VECS_PER_WARP WARP_SIZE/VEC_LENGTH
#define NVECS NWARPS*VECS_PER_WARP
//The padding alignment is to avoid bank conflict
#define LOW_ONCHIP_BUFFERSIZE 32+VEC_LENGTH
__global__ void kernel_lowLocality(TYPE *mat, TYPE *val, unsigned long num_thread_blocks,unsigned long num_segments, unsigned long num_elements, int ntimes)
{
    static __shared__ TYPE psum[32];
    int warp_lane = threadIdx.x % WARP_SIZE;
    int warp_id= threadIdx.x/WARP_SIZE;
    int lane = threadIdx.x % VEC_LENGTH;
    int wid= threadIdx.x/VEC_LENGTH;
    unsigned long dataBlockSize= num_segments*num_elements;
    unsigned long gidx = blockIdx.x*dataBlockSize;
#if TEST_SCRATCHPAD==1
    __shared__ TYPE data[NVECS][LOW_ONCHIP_BUFFERSIZE];
    for (int i= lane; i < LOW_ONCHIP_BUFFERSIZE; i+=VEC_LENGTH)
	data[wid][i]=0;
#endif

    TYPE tmp_val=0;
#if TEST_RANDOMICITY==1
    for(unsigned long j0=wid; j0<num_segments; j0+=NVECS)
    {
        unsigned long j= (j0+ wid*NVECS)%num_segments;
#else
    for(unsigned long j=wid; j<num_segments; j+=NVECS)
    {
#endif
        unsigned long start=gidx+j*num_elements;
        unsigned long stop= start+num_elements;
        #if TEST_SCRATCHPAD==1
        for (unsigned long i = start; i < stop; i+=LOW_ONCHIP_BUFFERSIZE) {
            //load part of the segment from HBM to scratchpad
            for (unsigned long i0 = lane; i0 < LOW_ONCHIP_BUFFERSIZE && i+i0<stop; i0+=VEC_LENGTH)
                data[wid][i0]= mat[i+i0];
	    if(num_elements<LOW_ONCHIP_BUFFERSIZE){
                for(int r=0; r<ntimes; r++)
                    for (unsigned long i0 = lane; i0 < num_elements; i0+=VEC_LENGTH)
                        data[wid][i0]+=1.0;
	    }else{
                for(int r=0; r<ntimes; r++)
                    for (unsigned long i0 = lane; i0 < LOW_ONCHIP_BUFFERSIZE; i0+=VEC_LENGTH)
                        data[wid][i0]+=1.0;
	    }
            //store the data back to HBM
            for (unsigned long i0 = lane; i0 < LOW_ONCHIP_BUFFERSIZE && i+i0<stop; i0+=VEC_LENGTH)
	    {
                mat[i+i0]= data[wid][i0];
	    }
        }
	#else
        for(int r=0; r<ntimes; r++)
           for (unsigned long i = start+ lane; i < stop; i+=VEC_LENGTH) {
	        #if ENABLE_READ==1
                tmp_val += mat[i];
	        #elif ENABLE_READWRITE==1
                tmp_val =mat[i]+1.0;
                mat[i]= tmp_val;
	        #endif
           }
	#endif
    }
#if ENABLE_READ==1
    tmp_val = warpSumReduction(tmp_val);
    if (warp_lane==0) psum[warp_id]=tmp_val;
    __syncthreads();
    tmp_val = (threadIdx.x < blockDim.x / WARP_SIZE) ? psum[warp_lane] : 0;
    if (warp_id==0)
    {
        tmp_val = warpSumReduction(tmp_val);
        if(threadIdx.x==0)atomicAdd(val, tmp_val);
    }
#elif ENABLE_READWRITE==1
#endif
}


#define LARGE_VEC_LENGTH 64
#define N_LARGE_VECS NWARPS*WARP_SIZE/LARGE_VEC_LENGTH
#define HIGH_ONCHIP_BUFFERSIZE 1024

__global__ void kernel_extremeLocality(TYPE *mat, TYPE *val, unsigned long num_thread_blocks,unsigned long num_segments, unsigned long num_elements, int ntimes)
{
    static __shared__ TYPE psum[32];
    int lane = threadIdx.x % LARGE_VEC_LENGTH;
    int wid = threadIdx.x / LARGE_VEC_LENGTH;
    int dataBlockSize= num_segments*num_elements;
    int gidx = blockIdx.x*dataBlockSize;
#if TEST_SCRATCHPAD==1
    __shared__ TYPE data[N_LARGE_VECS][HIGH_ONCHIP_BUFFERSIZE];
    for (int i= lane; i < HIGH_ONCHIP_BUFFERSIZE; i+=LARGE_VEC_LENGTH)
	data[wid][i]=0;
#endif


    TYPE tmp_val=0;
#if TEST_RANDOMICITY==1
    for(unsigned long j0=wid; j0<num_segments; j0+=N_LARGE_VECS)
    {
        unsigned long j= (j0+ wid*N_LARGE_VECS)%num_segments;
#else
    for(unsigned long j=wid; j<num_segments; j+=N_LARGE_VECS)
    {
#endif
        unsigned long start=gidx+j*num_elements;
        unsigned long stop= start+num_elements;
        #if TEST_SCRATCHPAD==1
        for (unsigned long i = start; i < stop; i+=ONCHIP_BUFFERSIZE) {
            //load part of the segment from HBM to scratchpad
            for (unsigned long i0 = lane; i0 < HIGH_ONCHIP_BUFFERSIZE && i+i0<stop; i0+=LARGE_VEC_LENGTH)
                data[wid][i0]= mat[i+i0];
            for(int r=0; r<ntimes; r++){
                for (unsigned long i0 = lane; i0 < HIGH_ONCHIP_BUFFERSIZE; i0+=LARGE_VEC_LENGTH){
                    data[wid][i0]+=1.0;
                }
            }
            //store the data back to HBM
            for (unsigned long i0 = lane; i0 < HIGH_ONCHIP_BUFFERSIZE && i+i0<stop; i0+=LARGE_VEC_LENGTH)
		mat[i+i0]= data[wid][i0];
        }
        #else
        for(int r=0; r<ntimes; r++)
           for (unsigned long i = start+ lane; i < stop; i+=LARGE_VEC_LENGTH) {
	    #if ENABLE_READ==1
               tmp_val += mat[i];
	       #elif ENABLE_READWRITE==1
               tmp_val =mat[i]+1.0;
               mat[i]= tmp_val;
	    #endif
           }
	#endif
    }
    #if ENABLE_READ==1
    tmp_val = warpSumReduction(tmp_val);
    if (lane==0) psum[wid]=tmp_val;
    __syncthreads();
    tmp_val = (threadIdx.x < blockDim.x / WARP_SIZE) ? psum[lane] : 0;
    if (wid==0){
        tmp_val = warpSumReduction(tmp_val);
        if(threadIdx.x==0)atomicAdd(val, tmp_val);
    }
#elif ENABLE_READWRITE==1
#endif
}


void cudaKernel(TYPE *d_mat, TYPE *d_val, unsigned long num_thread_blocks, unsigned long num_segments, unsigned long num_elements, int ntimes)
{
    if(num_elements<=256){
	std::cout<<"Low Locality: Breaking each warp to subwarps"<< std::endl;
	kernel_lowLocality<<< num_thread_blocks, THREADS_PER_BLOCK >>>(d_mat, d_val, num_thread_blocks, num_segments, num_elements, ntimes);
    }
    else{
	if(num_segments<32){
	    std::cout<<"High Locality: Breaking each warp to subwarps"<< std::endl;
	    kernel_extremeLocality<<< num_thread_blocks, THREADS_PER_BLOCK >>>(d_mat, d_val, num_thread_blocks, num_segments, num_elements, ntimes);
	}else kernel<<< num_thread_blocks, THREADS_PER_BLOCK >>>(d_mat, d_val, num_thread_blocks, num_segments, num_elements, ntimes);
    }
    hipDeviceSynchronize();

#ifdef DEBUG
    TYPE *val= new TYPE;
    cudaMemcpy(val, d_val, 1 * sizeof(TYPE), cudaMemcpyDeviceToHost);
    std::cout<<"Final sum"<< *val<<std::endl;
    free(val);
#endif
}
