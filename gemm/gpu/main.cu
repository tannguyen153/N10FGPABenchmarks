#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <vector>
using namespace std;
volatile int flag;
#include <omp.h>
#ifdef USE_NVML
  #include <nvml.h>
#endif
#include <unistd.h>
#define BLOCK_SIZE 128
#ifdef REPS
#define REPEATS REPS
#else 
#define REPEATS 10
#endif

#ifdef USE_NVML
static inline double GetPowerGPU(int DeviceID)
{       
    nvmlDevice_t device;
    nvmlReturn_t error;
    unsigned int power;
    size_t count = 0, powerSum = 0;

    // Initialize NVML library
    error = nvmlInit();  
    if (error != NVML_SUCCESS)
    {       
	printf("Failed to initialize NVML API with error code \"%s\".\n", nvmlErrorString(error));
#pragma omp barrier
	return -1;
    }

    // Get device handle
    error = nvmlDeviceGetHandleByIndex(DeviceID , &device);
    if (error != NVML_SUCCESS)
    {       
	printf("Failed to get device handle with error code \"%s\".\n", nvmlErrorString(error));
#pragma omp barrier
	return -1;
    }

#pragma omp barrier
    while(flag == 0)
    {       
	// Returns device power usage in mWatt
	error = nvmlDeviceGetPowerUsage(device, &power);
	if(error != NVML_SUCCESS)
	{
	    printf("Failed to get device power usage with error code \"%s\".\n", nvmlErrorString(error));
	    return -1;
	}
	powerSum = powerSum + power;
	count++;
	usleep(10);
    }
#pragma omp barrier

    error = nvmlShutdown();
    if (error != NVML_SUCCESS)
    {
	printf("Failed to shutdown NVML API with error code \"%s\".\n", nvmlErrorString(error));
	return -1;
    }

    return (double)(powerSum)/(double)(count * 1000.0); // Wattage is in mWatt, hence the division by 1000
}
#endif

#include <sys/time.h>
#include <stdio.h>
const double kMicro = 1.0e-6;
extern double getTime()
{
    struct timeval TV;

    const int RC = gettimeofday(&TV, NULL);
    if(RC == -1)
    {
	printf("ERROR: Bad call to gettimeofday\n");
	return(-1);
    }

    return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}  // end getTime()

/////
//KERNEL
/////
#define WPT_Y 8             // The work-per-thread
#define WPT_X 8            // The work-per-thread
#define RTS_Y (BLOCK_SIZE/WPT_Y)        // The reduced tile-size
#define RTS_X (BLOCK_SIZE/WPT_X)        // The reduced tile-size
#define TK    32
#define nLoads ((BLOCK_SIZE*TK)/(RTS_X*RTS_Y))

__global__ void  matrixMult( float  *C, float  *A, float *B, int A_width, int B_width)
{
    int K= A_width;
    // Local storage for a block of input matrices A and B
    __shared__ float  A_local[TK][BLOCK_SIZE];
    __shared__ float  B_local[TK][BLOCK_SIZE];
    // Block index
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;

    // Local ID index (offset within a block)
    int local_x = threadIdx.x;
    int local_y = threadIdx.y;
    float  acc[WPT_Y][WPT_X];
    //float  Breg[WPT_X];
    for (int wm=0; wm<WPT_Y; wm++) {
	for (int wn=0; wn<WPT_X; wn++) {
	    acc[wm][wn] = 0.0f;
	}
    }

    int numKtiles= K/TK;

    for(int t= 0; t<numKtiles; t++){ 
	for(int l=0; l<nLoads; l++){
	    int tid =local_y*RTS_Y + local_x;
	    int id = l*RTS_Y*RTS_X + tid;
	    int row = id%BLOCK_SIZE;
	    int col = id/BLOCK_SIZE;
	    int tiledIndex = TK*t + col;
	    A_local[col][row] = A[tiledIndex + A_width * (BLOCK_SIZE * block_y + row)];
	    B_local[col][row] = B[tiledIndex * B_width + (BLOCK_SIZE * block_x + row)];
	}
	__syncthreads();
	for (int k = 0; k < TK; k++)
	{
	    for (int wm=0; wm<WPT_Y; wm++) {
		float Areg = A_local[k][local_y + wm*RTS_Y];
		{
#if WPT_X==4
		    acc[wm][0] += Areg * B_local[h][k][local_x + 0*RTS_X];
		    acc[wm][1] += Areg * B_local[h][k][local_x + 1*RTS_X];
		    acc[wm][2] += Areg * B_local[h][k][local_x + 2*RTS_X];
		    acc[wm][3] += Areg * B_local[h][k][local_x + 3*RTS_X];
#elif WPT_X==8
		    acc[wm][0] += Areg * B_local[k][local_x + 0*RTS_X];
		    acc[wm][1] += Areg * B_local[k][local_x + 1*RTS_X];
		    acc[wm][2] += Areg * B_local[k][local_x + 2*RTS_X];
		    acc[wm][3] += Areg * B_local[k][local_x + 3*RTS_X];
		    acc[wm][4] += Areg * B_local[k][local_x + 4*RTS_X];
		    acc[wm][5] += Areg * B_local[k][local_x + 5*RTS_X];
		    acc[wm][6] += Areg * B_local[k][local_x + 6*RTS_X];
		    acc[wm][7] += Areg * B_local[k][local_x + 7*RTS_X];
#endif
		}
	    }
	}
	__syncthreads();
    }

    // Store result in matrix C
    int blockOffset= block_y *BLOCK_SIZE* A_width + block_x*BLOCK_SIZE;
    for (int wm=0; wm<WPT_Y; wm++) {
	int baseOffset= blockOffset+ (local_y+wm*RTS_Y)*A_width + local_x;
	for (int wn=0; wn<WPT_X; wn++) 
	{
	    C[baseOffset +  wn*RTS_X] = acc[wm][wn];
	}
    }
}


float* input_a_buf;
float* input_b_buf;
float* output_buf;

// Problem data.
int A_height = 32 * BLOCK_SIZE;
int A_width  = 16 * BLOCK_SIZE;
const int &B_height = A_width;
int B_width  = 16 * BLOCK_SIZE;
const int &C_height = A_height;
const int &C_width  = B_width;

float* input_a; 
float* input_b;
float* output; 
float* ref_output;

// Function prototypes
float rand_float();
bool init_cuda();
void init_problem();
void run(bool warmup, int repeats);
void compute_reference();
void verify();
void cleanup();

// Entry point.
int main(int argc, char **argv) {
    A_height = atoi(argv[1]);
    A_width  = atoi(argv[2]);
    B_width  = atoi(argv[3]);

    printf("Matrix sizes:\n  A: %d x %d\n  B: %d x %d\n  C: %d x %d\n",
	    A_height, A_width, B_height, B_width, C_height, C_width);

    // Spot check matrix sizes. They all must be a multiple of BLOCK_SIZE,
    // although it is relatively straightforward to handle non-multiples
    // by adding padding. For simplicity, this example does not pad.
    if((A_height % BLOCK_SIZE) != 0 || (A_width % BLOCK_SIZE) != 0 ||
	    (B_height % BLOCK_SIZE) != 0 || (B_width % BLOCK_SIZE) != 0 ||
	    (C_height % BLOCK_SIZE) != 0 || (C_width % BLOCK_SIZE) != 0) {
	printf("Matrix sizes must be a multiple of %d.\n", BLOCK_SIZE);
	return -1;
    }

    // Initialize OpenCL.
    hipSetDevice(0);
    if(!init_cuda()) {
	return -1;
    }
    // Initialize the problem data.
    init_problem();

    run(false, 1); //warm up
    // Run the kernel.
    run(true, REPEATS);

    // Free the resources allocated
    cleanup();
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
	printf("CUDA error: %s\n", hipGetErrorString(error));
	exit(-1);
    }

    return 0;
}

/////// HELPER FUNCTIONS ///////

// Randomly generate a floating-point number between -10 and 10.
float rand_float() {
    return float(rand()) / float(RAND_MAX) * 20.0f - 10.0f;
}

// Initializes device objects.
bool init_cuda() {
    printf("Initializing CUDA\n");
    hipMalloc(&input_a_buf, A_height * A_width * sizeof(float));
    hipMalloc(&input_b_buf, B_height * B_width * sizeof(float));
    hipMalloc(&output_buf, C_height * C_width * sizeof(float));
    return true;
}

// Initialize the data for the problem. 
void init_problem() {
    input_a= new float[A_height * A_width];
    input_b= new float[B_height * B_width];
    output= new float[C_height * C_width];
    printf("Generating input matrices\n");
    for(int j = 0; j < A_height * A_width; ++j) {
	input_a[j] = rand_float();
    }
    for(int i = 0; i < B_height * B_width; ++i) {
	input_b[i] = rand_float();
    }
}

void run(bool warmup, int repeats) {
    int A_size= A_height * A_width * sizeof(float);
    int B_size= B_height * B_width * sizeof(float);
    hipMemcpy(input_a_buf, input_a, A_size, hipMemcpyHostToDevice);
    hipMemcpy(input_b_buf, input_b, B_size, hipMemcpyHostToDevice);

    dim3 blockDim(RTS_X, RTS_Y, 1);
    dim3 gridDim((C_width/BLOCK_SIZE), (C_height/BLOCK_SIZE), 1);
    if(!warmup) printf("Dim block %d %d %d and grid %d %d", blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y);
    flag=0;
    const double start_time = getTime();

#if USE_NVML  
    double power;
#pragma omp parallel num_threads(2) shared(flag)        
    {       
	if (omp_get_thread_num() == 1)
	{       
	    power = GetPowerGPU(0);
	}
	else
	{
#pragma omp barrier
for(int r=0; r<REPEATS; r++) 
{
    cudaDeviceSynchronize();
    matrixMult<<< gridDim, blockDim >>>(output_buf, input_a_buf, input_b_buf, B_width, A_width);
    cudaDeviceSynchronize();
}
    	    flag=1;
#pragma omp barrier
	}
    }
    double end_time = getTime();
    const double total_time = (end_time - start_time)/REPEATS;
if(!warmup)
{
    printf("Average power consumption is %0.3lf watts\n", power);
}
#else

for(int r=0; r<REPEATS; r++) 
{
    hipDeviceSynchronize();
    matrixMult<<< gridDim, blockDim >>>(output_buf, input_a_buf, input_b_buf, B_width, A_width);
    hipDeviceSynchronize();
}
    double end_time = getTime();
    const double total_time = (end_time - start_time)/REPEATS;
#endif

if(!warmup)
{
    // Wall-clock time taken.
    printf("\nTime: %0.3f ms\n", total_time * 1e3);

    // Compute the throughput (GFLOPS).
    const size_t flops = (float)(2.0f * C_width * C_height * A_width / total_time);
    printf("\nThroughput: %0.2f GFLOPS\n\n", flops * 1e-9);
}

    // Read the result.
    hipMemcpy(output, output_buf, C_height * C_width * sizeof(float), hipMemcpyDeviceToHost);

    // Verify results.
#ifdef VERIFY
    compute_reference();
    verify();
#endif
}

void compute_reference() {
    // Compute the reference output.
    ref_output= new float[C_height * C_width];

    for(int y = 0; y < C_height; ++y) {
	for(int x = 0; x < C_width; ++x) {
	    // Compute result for C(y, x)
	    float sum = 0.0f;
	    for(int k = 0; k < A_width; ++k) {
		sum += input_a[y * A_width + k] * input_b[k * B_width + x];
	    }
	    ref_output[y * C_width + x] = sum;
	}
    }
    printf("Computing reference output\n");
}

void verify() {
    printf("Verifying\n");
    bool pass = true;
    for(int y = 0; y < C_height; ++y) {
	for(int x = 0; x < C_width; ++x) {
	    const float o = output[y * C_width + x];
	    const float r = ref_output[y * C_width + x];
	    const float d = o - r;
	    if(d*d > 1e-3){ pass=false;
		//printf("Failed at x y %d %d and values are %f %f\n", x, y, o, r);
	    }
	}
    }
    free(ref_output);
    printf("Verification: %s\n", pass ? "PASS" : "FAIL");
}

// Free the resources allocated during initialization
void cleanup() {
    if(input_a_buf) {
	hipFree(input_a_buf);
    }
    if(input_b_buf) {
	hipFree(input_b_buf);
    }
    if(output_buf) {
	hipFree(output_buf);
    }
    free(input_a);
    free(output);
    free(input_b);
}
